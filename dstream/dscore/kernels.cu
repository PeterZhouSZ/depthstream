#include "hip/hip_runtime.h"
#include "kernels.cuh"

//#define KERN_DEB

/////////////////////////////////////////////////////////////////////////////Helpers/////////////////////////////////////////////////////////////////////////////

#ifdef KERN_DEB
#include <iostream>
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}
#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)
#endif

#define DIVIDE_UP(a, b) (int)std::ceil((float)a / (float)b)
__forceinline__ __device__ unsigned long long int int2_to_ll(uint2 i){ return  __double_as_longlong(__hiloint2double(i.x, i.y)); }
__forceinline__ __device__ unsigned int uchar3_max_diff(uchar3 a, uchar3 b){ return max(abs(a.x - b.x), max(abs(a.y - b.y), abs(a.z - b.z))); }
__forceinline__ __device__ float4 add_float4(float4 a, float4 b) { return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w); }
__forceinline__ __device__ float4 subtract_float4(float4 a, float4 b) { return make_float4(a.x - b.x, a.y - b.y, a.z - b.z, a.w - b.w); }

/////////////////////////////////////////////////////////////////////////////Kernels/////////////////////////////////////////////////////////////////////////////

__global__
void color2gray_kernel(hipTextureObject_t input, unsigned char *output, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	int channels = 3;

	if (image_col < width && image_row < height){

		int b_pix = tex2D<unsigned char>(input, image_col * channels + 0, image_row);
		int g_pix = tex2D<unsigned char>(input, image_col * channels + 1, image_row);
		int r_pix = tex2D<unsigned char>(input, image_col * channels + 2, image_row);

		int gray = (int)((b_pix + g_pix + r_pix) / 3.0f);
		output[image_col + image_row * width] = (unsigned char)gray;
	}
}

__global__
void census_transform_kernel(hipTextureObject_t input_im, uint2 *output_census, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){
		unsigned char ref = tex2D<unsigned char>(input_im, image_col, image_row);
		/*unsigned int sum1 = 0, sum2 = 0;
		
		sum1 =
			((tex2D<unsigned char>(input_im, image_col, image_row - 2) > ref) << 31) +
			((tex2D<unsigned char>(input_im, image_col, image_row - 1) > ref) << 30) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row) > ref) << 29) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row) > ref) << 28) +
			((tex2D<unsigned char>(input_im, image_col, image_row + 1) > ref) << 27) +
			((tex2D<unsigned char>(input_im, image_col, image_row + 2) > ref) << 26);

			*/
		unsigned int sum1 = 0x0000;
		sum1 =
			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 4) > ref) << 31) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 4) > ref) << 30) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 4) > ref) << 29) +
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 4) > ref) << 28) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 4) > ref) << 27) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 4) > ref) << 26) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 4) > ref) << 25) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 3) > ref) << 24) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 3) > ref) << 23) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 3) > ref) << 22) +
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 3) > ref) << 21) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 3) > ref) << 20) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 3) > ref) << 19) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 3) > ref) << 18) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 2) > ref) << 17) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 2) > ref) << 16) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 2) > ref) << 15) +
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 2) > ref) << 14) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 2) > ref) << 13) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 2) > ref) << 12) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 2) > ref) << 11) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 1) > ref) << 10) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 1) > ref) << 9) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 1) > ref) << 8) +
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 1) > ref) << 7) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 1) > ref) << 6) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 1) > ref) << 5) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 1) > ref) << 4) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 0) > ref) << 3) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 0) > ref) << 2) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 0) > ref) << 1) +
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 0) > ref) << 0);

		unsigned int sum2 = 0x0000;
		sum2 =
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 0) > ref) << 31) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 0) > ref) << 30) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 0) > ref) << 29) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 0) > ref) << 28) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 1) > ref) << 27) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 1) > ref) << 26) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 1) > ref) << 25) +
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 1) > ref) << 24) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 1) > ref) << 23) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 1) > ref) << 22) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 1) > ref) << 21) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 2) > ref) << 20) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 2) > ref) << 19) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 2) > ref) << 18) +
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 2) > ref) << 17) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 2) > ref) << 16) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 2) > ref) << 15) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 2) > ref) << 14) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 3) > ref) << 13) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 3) > ref) << 12) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 3) > ref) << 11) +
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 3) > ref) << 10) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 3) > ref) << 9) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 3) > ref) << 8) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 3) > ref) << 7) +

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 4) > ref) << 6) +
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 4) > ref) << 5) +
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 4) > ref) << 4) +
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 4) > ref) << 3) +
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 4) > ref) << 2) +
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 4) > ref) << 1) +
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 4) > ref) << 0);

		output_census[image_row * width + image_col] = make_uint2(sum1, sum2);
	}
}

__global__
void cross_construct_kernel(hipTextureObject_t input_im, uchar4 *ouput_arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height){
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (image_row < height && image_col < width){

		uchar4 pix_arm = make_uchar4(0, 0, 0, 0);

		int ref = tex2D<unsigned char>(input_im, image_col, image_row);
		int scan_length, diff_curr_ref, diff_curr_next;

		//Upward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row - scan_length));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row - scan_length - 1));

			if (!(scan_length < max_arm_length &&
				image_row - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.x = scan_length;

		//Downward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row + scan_length));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row + scan_length + 1));

			if (!(scan_length < max_arm_length &&
				image_row + scan_length < height - 1 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.y = scan_length;

		//Leftward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col - scan_length, image_row));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col - scan_length - 1, image_row));

			if (!(scan_length < max_arm_length &&
				image_col - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.z = scan_length;

		//Rightward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col + scan_length, image_row));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col + scan_length + 1, image_row));

			if (!(scan_length < max_arm_length &&
				image_col + scan_length < width &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}
		pix_arm.w = scan_length;

		pix_arm.x = pix_arm.x == 0 ? (image_row - 2 >= 0 ? 2 : 0) : pix_arm.x;
		pix_arm.y = pix_arm.y == 0 ? (image_row + 2 < height ? 2 : 0) : pix_arm.y;
		pix_arm.z = pix_arm.z == 0 ? (image_col - 2 >= 0 ? 2 : 0) : pix_arm.z;
		pix_arm.w = pix_arm.w == 0 ? (image_col + 2 < width ? 2 : 0) : pix_arm.w;

		ouput_arm_vol[image_row * width + image_col] = pix_arm;
	}
}

__global__
void cross_construct_kernel_bgr(hipTextureObject_t input_im, uchar4 *ouput_arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height){
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;

	int channels = 3;

	if (image_row < height && image_col < width){

		uchar4 pix_arm = make_uchar4(0, 0, 0, 0);

		uchar3 ref = make_uchar3(
			tex2D<unsigned char>(input_im, image_col * channels + 0, image_row),
			tex2D<unsigned char>(input_im, image_col * channels + 1, image_row),
			tex2D<unsigned char>(input_im, image_col * channels + 2, image_row)
			);

		int scan_length, diff_curr_ref, diff_curr_next;

		//Upward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			uchar3 curr = make_uchar3(
				tex2D<unsigned char>(input_im, image_col * channels + 0, image_row - scan_length),
				tex2D<unsigned char>(input_im, image_col * channels + 1, image_row - scan_length),
				tex2D<unsigned char>(input_im, image_col * channels + 2, image_row - scan_length)
				);

			uchar3 next = make_uchar3(
				tex2D<unsigned char>(input_im, image_col * channels + 0, image_row - scan_length - 1),
				tex2D<unsigned char>(input_im, image_col * channels + 1, image_row - scan_length - 1),
				tex2D<unsigned char>(input_im, image_col * channels + 2, image_row - scan_length - 1)
				);

			diff_curr_ref = uchar3_max_diff(curr, ref);
			diff_curr_next = uchar3_max_diff(curr, next);

			if (!(scan_length < max_arm_length &&
				image_row - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.x = scan_length;

		//Downward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			uchar3 curr = make_uchar3(
				tex2D<unsigned char>(input_im, image_col * channels + 0, image_row + scan_length),
				tex2D<unsigned char>(input_im, image_col * channels + 1, image_row + scan_length),
				tex2D<unsigned char>(input_im, image_col * channels + 2, image_row + scan_length)
				);

			uchar3 next = make_uchar3(
				tex2D<unsigned char>(input_im, image_col * channels + 0, image_row + scan_length + 1),
				tex2D<unsigned char>(input_im, image_col * channels + 1, image_row + scan_length + 1),
				tex2D<unsigned char>(input_im, image_col * channels + 2, image_row + scan_length + 1)
				);

			diff_curr_ref = uchar3_max_diff(curr, ref);
			diff_curr_next = uchar3_max_diff(curr, next);

			if (!(scan_length < max_arm_length &&
				image_row + scan_length < height - 1 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.y = scan_length;

		//Leftward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			uchar3 curr = make_uchar3(
				tex2D<unsigned char>(input_im, (image_col - scan_length) * channels + 0, image_row),
				tex2D<unsigned char>(input_im, (image_col - scan_length) * channels + 1, image_row),
				tex2D<unsigned char>(input_im, (image_col - scan_length) * channels + 2, image_row)
				);

			uchar3 next = make_uchar3(
				tex2D<unsigned char>(input_im, (image_col - scan_length - 1) * channels + 0, image_row),
				tex2D<unsigned char>(input_im, (image_col - scan_length - 1) * channels + 1, image_row),
				tex2D<unsigned char>(input_im, (image_col - scan_length - 1) * channels + 2, image_row)
				);

			diff_curr_ref = uchar3_max_diff(curr, ref);
			diff_curr_next = uchar3_max_diff(curr, next);

			if (!(scan_length < max_arm_length &&
				image_col - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.z = scan_length;

		//Rightward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			uchar3 curr = make_uchar3(
				tex2D<unsigned char>(input_im, (image_col + scan_length) * channels + 0, image_row),
				tex2D<unsigned char>(input_im, (image_col + scan_length) * channels + 1, image_row),
				tex2D<unsigned char>(input_im, (image_col + scan_length) * channels + 2, image_row)
				);

			uchar3 next = make_uchar3(
				tex2D<unsigned char>(input_im, (image_col + scan_length + 1) * channels + 0, image_row),
				tex2D<unsigned char>(input_im, (image_col + scan_length + 1) * channels + 1, image_row),
				tex2D<unsigned char>(input_im, (image_col + scan_length + 1) * channels + 2, image_row)
				);

			diff_curr_ref = uchar3_max_diff(curr, ref);
			diff_curr_next = uchar3_max_diff(curr, next);

			if (!(scan_length < max_arm_length &&
				image_col + scan_length < width &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}
		pix_arm.w = scan_length;

		pix_arm.x = pix_arm.x == 0 ? (image_row - 2 >= 0 ? 2 : 0) : pix_arm.x;
		pix_arm.y = pix_arm.y == 0 ? (image_row + 2 < height ? 2 : 0) : pix_arm.y;
		pix_arm.z = pix_arm.z == 0 ? (image_col - 2 >= 0 ? 2 : 0) : pix_arm.z;
		pix_arm.w = pix_arm.w == 0 ? (image_col + 2 < width ? 2 : 0) : pix_arm.w;

		ouput_arm_vol[image_row * width + image_col] = pix_arm;
	}
}

__global__
void cost_initialization_kernel(hipTextureObject_t left_tex, hipTextureObject_t left_census_tex, hipTextureObject_t right_tex, hipTextureObject_t right_census_tex, float4 *cost_vol, float ad_gamma, float census_gamma, bool left_to_right, int width, int height){
	//16 x 16 threads
	extern __shared__ float4 cost_cache[];

	int image_row = blockIdx.y;
	float4 accum = { 0.0f, 0.0f, 0.0f, 0.0f };

	for (int col_iter = 0, col_iter_lim = DIVIDE_UP(width, blockDim.y); col_iter < col_iter_lim; col_iter++){

		int image_col = __mul24(col_iter, blockDim.y) + threadIdx.y;

		if (image_row < height && image_col < width){

			//Compute costs
			float4 ad_costs, census_costs;
			unsigned int disps[4] = { blockDim.x * 0 + threadIdx.x + 1, blockDim.x * 1 + threadIdx.x + 1, blockDim.x * 2 + threadIdx.x + 1, blockDim.x * 3 + threadIdx.x + 1 };

			if (left_to_right){
				ad_costs = make_float4(
					(fabsf(tex2D<unsigned char>(left_tex, image_col, image_row) - tex2D<unsigned char>(right_tex, image_col - disps[0], image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col, image_row) - tex2D<unsigned char>(right_tex, image_col - disps[1], image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col, image_row) - tex2D<unsigned char>(right_tex, image_col - disps[2], image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col, image_row) - tex2D<unsigned char>(right_tex, image_col - disps[3], image_row)) / 255.0f) * ad_gamma
					);
				census_costs = make_float4(
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col, image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col - disps[0], image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col, image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col - disps[1], image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col, image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col - disps[2], image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col, image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col - disps[3], image_row)))) / 6.0f) * census_gamma
					);
			}
			else{
				ad_costs = make_float4(
					(fabsf(tex2D<unsigned char>(left_tex, image_col + disps[0], image_row) - tex2D<unsigned char>(right_tex, image_col, image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col + disps[1], image_row) - tex2D<unsigned char>(right_tex, image_col, image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col + disps[2], image_row) - tex2D<unsigned char>(right_tex, image_col, image_row)) / 255.0f) * ad_gamma,
					(fabsf(tex2D<unsigned char>(left_tex, image_col + disps[3], image_row) - tex2D<unsigned char>(right_tex, image_col, image_row)) / 255.0f) * ad_gamma
					);
				census_costs = make_float4(
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col + disps[0], image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col, image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col + disps[1], image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col, image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col + disps[2], image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col, image_row)))) / 6.0f) * census_gamma,
					(__popcll((int2_to_ll(tex2D<uint2>(left_census_tex, image_col + disps[3], image_row)) ^ int2_to_ll(tex2D<uint2>(right_census_tex, image_col, image_row)))) / 6.0f) * census_gamma
					);
			}

			//Compute the total cost

			cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x] = add_float4(ad_costs, census_costs);
			__syncthreads();

			//Compute prefix sum
			float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			for (int i = 0; i <= threadIdx.y; i++){
				sum = add_float4(sum, cost_cache[__mul24(i, blockDim.x) + threadIdx.x]);
			}
			__syncthreads();

			//Write to global memory
			cost_vol[threadIdx.x + __mul24(image_col, blockDim.x) + __mul24(image_row, __mul24(width, blockDim.x))] = add_float4(sum, accum);

			//Update cache
			if (threadIdx.y == blockDim.y - 1) cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x] = sum;
			__syncthreads();

			//Update accumulator
			accum = add_float4(accum, cost_cache[__mul24(blockDim.y - 1, blockDim.x) + threadIdx.x]);
		}
	}
}

__global__
void horizontal_aggregation_kernel(float4 *cost_vol_in, uchar4 *arm_vol, float4 *cost_vol_out, int width, int height, int max_disparity){
	extern __shared__ float4 cost_cache[];

	int image_col = blockIdx.x;

	float4 accum = { 0.0f, 0.0f, 0.0f, 0.0f };

	int prev_right_lim = -1;
	int prev_left_lim = -1;
	float4 right_lim_temp;
	float4 left_lim_temp;

	for (int row_iter = 0, row_iter_lim = DIVIDE_UP(height, blockDim.y); row_iter < row_iter_lim; row_iter++){

		int image_row = __mul24(row_iter, blockDim.y) + threadIdx.y;

		if (image_row < height && image_col < width){

			//Load arm data
			uchar4 pix_arm = arm_vol[image_col + image_row * width];
			int right_lim = image_col + pix_arm.w; int left_lim = image_col - pix_arm.z - 1;

			//Aggregate
			float4 aggregate;
			if (prev_right_lim != right_lim){
				right_lim_temp = cost_vol_in[threadIdx.x + __mul24(right_lim, blockDim.x) + __mul24(image_row, __mul24(width, blockDim.x))];
				prev_right_lim = right_lim;
			}

			if (left_lim >= 0 && prev_left_lim != left_lim){
				left_lim_temp = cost_vol_in[threadIdx.x + __mul24(left_lim, blockDim.x) + __mul24(image_row, __mul24(width, blockDim.x))];
				prev_left_lim = left_lim;
				aggregate = subtract_float4(aggregate, left_lim_temp);
			}

			cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x] = aggregate;
			__syncthreads();

			//Compute prefix sum
			float4 sum = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
			for (int i = 0; i <= threadIdx.y; i++){ sum = add_float4(sum, cost_cache[__mul24(i, blockDim.x) + threadIdx.x]); }
			__syncthreads();

			//Write to global memory
			cost_vol_out[threadIdx.x + __mul24(image_col, blockDim.x) + __mul24(image_row, __mul24(width, blockDim.x))] = add_float4(sum, accum);

			//Update cache
			if (threadIdx.y == blockDim.y - 1) cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x] = sum;
			__syncthreads();

			//Update accumulator
			accum = add_float4(accum, cost_cache[__mul24(blockDim.y - 1, blockDim.x) + threadIdx.x]);
		}
	}
}

__global__
void vertical_aggregation_kernel(float4 *cost_vol_in, uchar4 *arm_vol, float4 *cost_vol_out, unsigned char *disp_im, int width, int height, int max_disparity){
	extern __shared__ float4 cost_cache[];

	int image_row = blockIdx.y;

	for (int col_iter = 0, col_iter_lim = DIVIDE_UP(width, blockDim.y); col_iter < col_iter_lim; col_iter++){

		int image_col = __mul24(col_iter, blockDim.y) + threadIdx.y;

		if (image_row < height && image_col < width){

			//Load arm data
			uchar4 pix_arm = arm_vol[image_col + __mul24(image_row, width)];

			int down_lim = image_row + pix_arm.y;
			int up_lim = image_row - pix_arm.x - 1;

			//Aggregate
			float4 aggregate = cost_vol_in[threadIdx.x + __mul24(image_col, blockDim.x) + __mul24(down_lim, __mul24(width, blockDim.x))];

			if (up_lim >= 0){
				float4 up_lim_temp = cost_vol_in[threadIdx.x + __mul24(image_col, blockDim.x) + __mul24(up_lim, __mul24(width, blockDim.x))];
				aggregate = subtract_float4(aggregate, up_lim_temp);
			}

			//Find the local minimum
			bool invalid = false;

			unsigned int disps[4] = { blockDim.x * 0 + threadIdx.x + 1, blockDim.x * 1 + threadIdx.x + 1, blockDim.x * 2 + threadIdx.x + 1, blockDim.x * 3 + threadIdx.x + 1 };
			float min_cost = aggregate.x; float min_disp = disps[0];

			if (aggregate.y < min_cost){ min_cost = aggregate.y; min_disp = disps[1]; }
			else if (aggregate.y == min_cost) { invalid = true; }
			else{}

			if (aggregate.z < min_cost){ min_cost = aggregate.z; min_disp = disps[2]; }
			else if (aggregate.z == min_cost) { invalid = true; }
			else{}

			if (aggregate.w < min_cost){ min_cost = aggregate.w; min_disp = disps[3]; }
			else if (aggregate.w == min_cost) { invalid = true; }
			else{}

			cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x = min_cost;
			cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].y = min_disp;
			__syncthreads();

			//Find the global minimum
			int thread_selector = 2; int stride = 1;

			for (int reduce_iter = 0, reduce_iter_lim = (int)ceil(log2f(blockDim.x)) - 1; reduce_iter < reduce_iter_lim; reduce_iter++){
				if (threadIdx.x % thread_selector == 0){
					if (((threadIdx.x + stride) >= blockDim.x ? INFINITY : cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x) < cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x){
						cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x = cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x;
						cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].y = cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].y;
					}
					else if (((threadIdx.x + stride) >= blockDim.x ? INFINITY : cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x) == cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x) invalid = true;
					else{}
				}
				__syncthreads();

				thread_selector = thread_selector << 1; stride = stride << 1;
			}

			if (threadIdx.x % thread_selector == 0){
				if (((threadIdx.x + stride) >= blockDim.x ? INFINITY : cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x) < cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x){
					cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x = cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x;
					cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].y = cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].y;
				}
				else if (((threadIdx.x + stride) >= blockDim.x ? INFINITY : cost_cache[__mul24(threadIdx.y, blockDim.x) + (threadIdx.x + stride)].x) == cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].x) invalid = true;
				else{}
				disp_im[image_col + __mul24(image_row, width)] = invalid ? OUTLIER : (char)cost_cache[__mul24(threadIdx.y, blockDim.x) + threadIdx.x].y;
			}
		}
	}
}

__global__
void consistency_check_kernel(hipTextureObject_t left_disp_im, hipTextureObject_t right_disp_im, unsigned char *output_disp_im, int disparity_tolerance, int width, int height){

	int col_to_access = blockIdx.x * blockDim.x + threadIdx.x;
	int row_to_access = blockIdx.y * blockDim.y + threadIdx.y;

	if (row_to_access < height && col_to_access < width){
		unsigned char disp = tex2D<unsigned char>(left_disp_im, col_to_access, row_to_access);
		unsigned char to_check = tex2D<unsigned char>(right_disp_im, col_to_access - disp, row_to_access);

		output_disp_im[row_to_access * width + col_to_access] = (abs(disp - to_check) <= disparity_tolerance) ? disp : OUTLIER;
	}
}

__global__
void horizontal_voting_kernel(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned char *output_disp, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){

		int sums[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
		int eligible_votes = 0;
		int no_of_votes = 0;

		//Load arm data
		uchar4 pix_arm = arm_vol[image_col + image_row * width];

		//Check disp value
		int disp_value = tex2D<unsigned char>(input_disp, image_col, image_row);

		if (disp_value == OUTLIER){
			for (int pix_iter = -pix_arm.z; pix_iter <= pix_arm.w; pix_iter++){
				int disp_val = tex2D<unsigned char>(input_disp, image_col + pix_iter, image_row);
				if (disp_val != OUTLIER){
					sums[0] += ((disp_val & 1) != 0);
					sums[1] += ((disp_val & 2) != 0);
					sums[2] += ((disp_val & 4) != 0);
					sums[3] += ((disp_val & 8) != 0);
					sums[4] += ((disp_val & 16) != 0);
					sums[5] += ((disp_val & 32) != 0);
					sums[6] += ((disp_val & 64) != 0);
					sums[7] += ((disp_val & 128) != 0);
					eligible_votes++;
				}
				no_of_votes++;
			}
			__syncthreads();

			int majority = eligible_votes * 0.5;
			disp_value = (((sums[7] > majority) << 7) +
				((sums[6] > majority) << 6) +
				((sums[5] > majority) << 5) +
				((sums[4] > majority) << 4) +
				((sums[3] > majority) << 3) +
				((sums[2] > majority) << 2) +
				((sums[1] > majority) << 1) +
				((sums[0] > majority) << 0));
			disp_value = (eligible_votes > no_of_votes * 0.65f) ? disp_value : OUTLIER;
		}

		output_disp[image_col + image_row * width] = disp_value;
	}//bounds check
}//kernel

__global__
void vertical_voting_kernel(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned char *output_disp, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){

		int sums[8] = { 0, 0, 0, 0, 0, 0, 0, 0 };
		int eligible_votes = 0;
		int no_of_votes = 0;

		//Load arm data
		uchar4 pix_arm = arm_vol[image_col + image_row * width];

		//Check disp value
		int disp_value = tex2D<unsigned char>(input_disp, image_col, image_row);

		if (disp_value == OUTLIER){
			for (int pix_iter = -pix_arm.x; pix_iter <= pix_arm.y; pix_iter++){
				int disp_val = tex2D<unsigned char>(input_disp, image_col, image_row + pix_iter);
				if (disp_val != OUTLIER){
					sums[0] += ((disp_val & 1) != 0);
					sums[1] += ((disp_val & 2) != 0);
					sums[2] += ((disp_val & 4) != 0);
					sums[3] += ((disp_val & 8) != 0);
					sums[4] += ((disp_val & 16) != 0);
					sums[5] += ((disp_val & 32) != 0);
					sums[6] += ((disp_val & 64) != 0);
					sums[7] += ((disp_val & 128) != 0);
					eligible_votes++;
				}
				no_of_votes++;
			}
			__syncthreads();

			int majority = eligible_votes * 0.5;
			disp_value = (((sums[7] > majority) << 7) +
				((sums[6] > majority) << 6) +
				((sums[5] > majority) << 5) +
				((sums[4] > majority) << 4) +
				((sums[3] > majority) << 3) +
				((sums[2] > majority) << 2) +
				((sums[1] > majority) << 1) +
				((sums[0] > majority) << 0));
			disp_value = (eligible_votes > no_of_votes * 0.65f) ? disp_value : OUTLIER;
		}

		output_disp[image_col + image_row * width] = disp_value;
	}//bounds check
}//kernel

__global__
void extrapolation_kernel(hipTextureObject_t input_disp, unsigned char *output_disp, int width, int height){
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (image_row < height && image_col < width){
		if (tex2D<unsigned char>(input_disp, image_col, image_row) == OUTLIER){
			int left_disp = OUTLIER, right_disp = OUTLIER;

			int gap = 0;
			bool is_left_edge = false, is_right_edge = false;

			//Scan leftward
			for (int i = image_col; i >= 0; i--){
				if (tex2D<unsigned char>(input_disp, i, image_row) == OUTLIER){
					gap++;
					continue;
				}
				else{
					is_left_edge = i <= 0;
					left_disp = tex2D<unsigned char>(input_disp, i, image_row);
					break;
				}
			}

			//Scan rightward
			for (int i = image_col; i < width; i++){
				if (tex2D<unsigned char>(input_disp, i, image_row) == OUTLIER){
					gap++;
					continue;
				}
				else{
					is_right_edge = i >= (width - 1);
					right_disp = tex2D<unsigned char>(input_disp, i, image_row);
					break;
				}

			}

			int val = min(is_left_edge ? right_disp : left_disp, is_right_edge ? left_disp : right_disp);
			val = gap <= 40 ? val : ((is_right_edge || is_left_edge) ? val : OUTLIER);
			output_disp[image_col + image_row * width] = val;
		}
	}
}

__global__
void cleanup_kernel(unsigned char *input_disp, unsigned char *output_disp, int width, int height, int min_disparity){
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (image_row < height && image_col < width){
		int disp = input_disp[image_col + image_row * width];
		if (disp == OUTLIER || disp < min_disparity){
			output_disp[image_col + image_row * width] = 0;
		}
	}
}


/////////////////////////////////////////////////////////////////////////////Stubs/////////////////////////////////////////////////////////////////////////////

void census_transform(hipTextureObject_t input_im, uint2 *output_census, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	census_transform_kernel << <blocks, threads >> >(input_im, output_census, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Census transform failed.");
#endif
}

void cross_construct(hipTextureObject_t input_im, uchar4 *arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	cross_construct_kernel << <blocks, threads >> >(input_im, arm_vol, arm_length, max_arm_length, arm_threshold, strict_arm_threshold, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Cross construct failed.");
#endif
}

void cross_construct_bgr(hipTextureObject_t input_im, uchar4 *arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	cross_construct_kernel_bgr << <blocks, threads >> >(input_im, arm_vol, arm_length, max_arm_length, arm_threshold, strict_arm_threshold, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Cross construct BGR failed.");
#endif
}

void match(hipTextureObject_t left_tex, hipTextureObject_t left_census_tex, hipTextureObject_t right_tex, hipTextureObject_t right_census_tex, float4 *cost_vol_temp_a, float4 *cost_vol_temp_b, uchar4 *arm_vol, unsigned char *disp_im, float ad_gamma, float census_gamma, bool left_to_right, int width, int height, int max_disparity, hipStream_t stream){

	dim3 threads;
	threads.x = max_disparity / GRANULARITY;
	threads.y = max_disparity >= 32 ? 16 : 32;

	dim3 blocks(1, height);

	size_t smem_sz = threads.x * threads.y * sizeof(float) * GRANULARITY;
	cost_initialization_kernel << <blocks, threads, smem_sz >> >(left_tex, left_census_tex, right_tex, right_census_tex, cost_vol_temp_a, ad_gamma, census_gamma, left_to_right, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Cost initialization failed.");
#endif

	blocks.x = width; blocks.y = 1;
	horizontal_aggregation_kernel << <blocks, threads, smem_sz >> >(cost_vol_temp_a, arm_vol, cost_vol_temp_b, width, height, max_disparity);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal aggregation failed.");
#endif

	blocks.x = 1; blocks.y = height;
	vertical_aggregation_kernel << <blocks, threads, smem_sz >> >(cost_vol_temp_b, arm_vol, cost_vol_temp_a, disp_im, width, height, max_disparity);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal aggregation failed.");
#endif
}

void check_consistency(hipTextureObject_t left_disp_im, hipTextureObject_t right_disp_im, unsigned char *output_disp_im, int disparity_tolerance, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	consistency_check_kernel << <blocks, threads >> >(left_disp_im, right_disp_im, output_disp_im, disparity_tolerance, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Consistency check failed.");
#endif
}

void horizontal_voting(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned char *output_disp, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	horizontal_voting_kernel << <blocks, threads >> >(input_disp, arm_vol, output_disp, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal voting failed.");
#endif
}

void vertical_voting(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned char *output_disp, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	vertical_voting_kernel << <blocks, threads >> >(input_disp, arm_vol, output_disp, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Vertical voting failed.");
#endif
}

void extrapolation(hipTextureObject_t input_disp, unsigned char *output_disp, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	extrapolation_kernel << <blocks, threads >> >(input_disp, output_disp, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Extrapolation failed.");
#endif
}

void color2gray(hipTextureObject_t input, unsigned char *output, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	color2gray_kernel << <blocks, threads >> >(input, output, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Color conversion failed.");
#endif
}

void cleanup(unsigned char *input, unsigned char *output, int width, int height, int min_disparity, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	cleanup_kernel << <blocks, threads >> >(input, output, width, height, min_disparity);
}