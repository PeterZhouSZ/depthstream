#include "hip/hip_runtime.h"
#include "DSKernels.cuh"
#include <opencv2\opencv.hpp>

//#define KERN_DEB

#define BLOCK_X  16
#define BLOCK_Y  16

// Exchange trick: Morgan McGuire, ShaderX 2008
#define s2(a,b)            { unsigned short tmp = a; a = min(a,b); b = max(tmp,b); }
#define mn3(a,b,c)         s2(a,b); s2(a,c);
#define mx3(a,b,c)         s2(b,c); s2(a,c);

#define mnmx3(a,b,c)       mx3(a,b,c); s2(a,b);                               // 3 exchanges
#define mnmx4(a,b,c,d)     s2(a,b); s2(c,d); s2(a,c); s2(b,d);                // 4 exchanges
#define mnmx5(a,b,c,d,e)   s2(a,b); s2(c,d); mn3(a,c,e); mx3(b,d,e);          // 6 exchanges
#define mnmx6(a,b,c,d,e,f) s2(a,d); s2(b,e); s2(c,f); mn3(a,b,c); mx3(d,e,f); // 7 exchanges

#define SMEM(x,y)  smem[(x)+1][(y)+1]
#define IN(x,y)    d_in[(y)*nx + (x)]

/////////////////////////////////////////////////////////////////////////////Helpers/////////////////////////////////////////////////////////////////////////////

#ifdef KERN_DEB
#include <iostream>
static inline void _safe_cuda_call(hipError_t err, const char* msg, const char* file_name, const int line_number)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "%s\n\nFile: %s\n\nLine Number: %d\n\nReason: %s\n", msg, file_name, line_number, hipGetErrorString(err));
		std::cin.get();
		exit(EXIT_FAILURE);
	}
}
#define SAFE_CALL(call,msg) _safe_cuda_call((call),(msg),__FILE__,__LINE__)
#endif

#define DIVIDE_UP(a, b) (int)std::ceil((float)a / (float)b)

/////////////////////////////////////////////////////////////////////////////Kernels/////////////////////////////////////////////////////////////////////////////

__global__
void census_transform_kernel(hipTextureObject_t input_im, unsigned long long int *output_census, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){
		unsigned char ref = tex2D<unsigned char>(input_im, image_col, image_row);

		unsigned int sum1 = 0x0000;
		sum1 =
			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 4) > ref) << 31) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 4) > ref) << 30) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 4) > ref) << 29) |
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 4) > ref) << 28) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 4) > ref) << 27) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 4) > ref) << 26) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 4) > ref) << 25) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 3) > ref) << 24) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 3) > ref) << 23) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 3) > ref) << 22) |
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 3) > ref) << 21) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 3) > ref) << 20) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 3) > ref) << 19) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 3) > ref) << 18) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 2) > ref) << 17) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 2) > ref) << 16) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 2) > ref) << 15) |
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 2) > ref) << 14) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 2) > ref) << 13) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 2) > ref) << 12) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 2) > ref) << 11) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 1) > ref) << 10) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 1) > ref) << 9) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 1) > ref) << 8) |
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 1) > ref) << 7) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row - 1) > ref) << 6) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row - 1) > ref) << 5) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row - 1) > ref) << 4) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row - 0) > ref) << 3) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row - 0) > ref) << 2) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row - 0) > ref) << 1) |
			((tex2D<unsigned char>(input_im, image_col - 0, image_row - 0) > ref) << 0);

		unsigned int sum2 = 0x0000;
		sum2 =
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 0) > ref) << 31) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 0) > ref) << 30) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 0) > ref) << 29) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 0) > ref) << 28) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 1) > ref) << 27) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 1) > ref) << 26) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 1) > ref) << 25) |
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 1) > ref) << 24) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 1) > ref) << 23) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 1) > ref) << 22) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 1) > ref) << 21) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 2) > ref) << 20) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 2) > ref) << 19) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 2) > ref) << 18) |
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 2) > ref) << 17) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 2) > ref) << 16) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 2) > ref) << 15) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 2) > ref) << 14) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 3) > ref) << 13) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 3) > ref) << 12) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 3) > ref) << 11) |
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 3) > ref) << 10) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 3) > ref) << 9) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 3) > ref) << 8) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 3) > ref) << 7) |

			((tex2D<unsigned char>(input_im, image_col - 3, image_row + 4) > ref) << 6) |
			((tex2D<unsigned char>(input_im, image_col - 2, image_row + 4) > ref) << 5) |
			((tex2D<unsigned char>(input_im, image_col - 1, image_row + 4) > ref) << 4) |
			((tex2D<unsigned char>(input_im, image_col + 0, image_row + 4) > ref) << 3) |
			((tex2D<unsigned char>(input_im, image_col + 1, image_row + 4) > ref) << 2) |
			((tex2D<unsigned char>(input_im, image_col + 2, image_row + 4) > ref) << 1) |
			((tex2D<unsigned char>(input_im, image_col + 3, image_row + 4) > ref) << 0);

		uint2 temp = make_uint2(sum1, sum2);
		output_census[image_row * width + image_col] = *reinterpret_cast<unsigned long long int*>(&temp);
	}
}

__global__
void cross_construct_kernel(hipTextureObject_t input_im, uchar4 *ouput_arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height){
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;

	if (image_row < height && image_col < width){

		uchar4 pix_arm = make_uchar4(0, 0, 0, 0);

		int ref = tex2D<unsigned char>(input_im, image_col, image_row);
		int scan_length, diff_curr_ref, diff_curr_next;

		//Upward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row - scan_length));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row - scan_length - 1));

			if (!(scan_length < max_arm_length &&
				image_row - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.x = scan_length;

		//Downward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row + scan_length));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col, image_row + scan_length + 1));

			if (!(scan_length < max_arm_length &&
				image_row + scan_length < height - 1 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.y = scan_length;

		//Leftward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col - scan_length, image_row));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col - scan_length - 1, image_row));

			if (!(scan_length < max_arm_length &&
				image_col - scan_length > 0 &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}

		pix_arm.z = scan_length;

		//Rightward scan
		scan_length = 0; diff_curr_ref = 0; diff_curr_next = 0;
		while (true)
		{
			diff_curr_ref = abs(ref - tex2D<unsigned char>(input_im, image_col + scan_length, image_row));
			diff_curr_next = abs(ref - tex2D<unsigned char>(input_im, image_col + scan_length + 1, image_row));

			if (!(scan_length < max_arm_length &&
				image_col + scan_length < width &&
				diff_curr_ref <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold) &&
				diff_curr_next <= (arm_length < scan_length ? strict_arm_threshold : arm_threshold))) break;

			scan_length++;
		}
		pix_arm.w = scan_length;

		pix_arm.x = pix_arm.x == 0 ? (image_row - 2 >= 0 ? 2 : 0) : pix_arm.x;
		pix_arm.y = pix_arm.y == 0 ? (image_row + 2 < height ? 2 : 0) : pix_arm.y;

		//pix_arm.x = image_row - 2 >= 0 ? 2 : pix_arm.x;
		//pix_arm.y = image_row + 2 < height ? 2 : pix_arm.y;

		pix_arm.z = pix_arm.z == 0 ? (image_col - 2 >= 0 ? 2 : 0) : pix_arm.z;
		pix_arm.w = pix_arm.w == 0 ? (image_col + 2 < width ? 2 : 0) : pix_arm.w;

		ouput_arm_vol[image_row * width + image_col] = pix_arm;
	}
}

__global__
void cost_initialization_kernel(unsigned char *left, unsigned char *right, unsigned long long int *left_census, unsigned long long int *right_census, float *cost_vol, float ad_gamma, float census_gamma, bool left_to_right, int width, int height){
	extern __shared__ unsigned char temp[];

	unsigned char *ref_temp = temp;
	unsigned char *targ_temp = &ref_temp[blockDim.x];

	unsigned long long int *ref_census_temp = (unsigned long long int*)&targ_temp[blockDim.x * 2];
	unsigned long long int *targ_census_temp = &ref_census_temp[blockDim.x];

	//Initialize to zero
	ref_temp[threadIdx.x] = 0;
	targ_temp[threadIdx.x] = 0;
	targ_temp[blockDim.x + threadIdx.x] = 0;

	ref_census_temp[threadIdx.x] = 0;
	targ_census_temp[threadIdx.x] = 0;
	targ_census_temp[blockDim.x + threadIdx.x] = 0;

	__syncthreads();

	int image_row = blockIdx.y;

	float cost = 0.0f;

	if (image_row < height){

		if (left_to_right){
			for (int image_col = 0; image_col < width; image_col++){

				int block_index = image_col % blockDim.x;

				if (block_index == 0){
					if (image_col + threadIdx.x < width){
						ref_temp[threadIdx.x] = left[image_row * width + image_col + threadIdx.x];
						ref_census_temp[threadIdx.x] = left_census[image_row * width + image_col + threadIdx.x];
					}

					if (image_col + threadIdx.x < width){
						targ_temp[blockDim.x + threadIdx.x] = right[image_row * width + image_col + threadIdx.x];
						targ_census_temp[blockDim.x + threadIdx.x] = right_census[image_row * width + image_col + threadIdx.x];
					}

					if ((int)(image_col - blockDim.x + threadIdx.x) >= 0 && (int)(image_col - blockDim.x + threadIdx.x) < width){
						targ_temp[threadIdx.x] = right[image_row * width + image_col - blockDim.x + threadIdx.x];
						targ_census_temp[threadIdx.x] = right_census[image_row * width + image_col - blockDim.x + threadIdx.x];
					}
					__syncthreads();
				}

				float ad_cost, census_cost;

				ad_cost = (fabsf(ref_temp[block_index] - targ_temp[blockDim.x + block_index - threadIdx.x]) / 255.0f) * ad_gamma;
				census_cost = (__popcll(ref_census_temp[block_index] ^ targ_census_temp[blockDim.x + block_index - threadIdx.x]) / 64.0f) * census_gamma;

				cost += ad_cost + census_cost;

				cost_vol[image_row * width * blockDim.x + image_col * blockDim.x + threadIdx.x] = cost;
			}
		}
		else{

			for (int image_col = 0; image_col < width; image_col++){

				int block_index = image_col % blockDim.x;

				if (block_index == 0){

					if (image_col + threadIdx.x < width){
						ref_temp[threadIdx.x] = right[image_row * width + image_col + threadIdx.x];
						ref_census_temp[threadIdx.x] = right_census[image_row * width + image_col + threadIdx.x];
					}

					if (image_col + threadIdx.x < width){
						targ_temp[threadIdx.x] = left[image_row * width + image_col + threadIdx.x];
						targ_census_temp[threadIdx.x] = left_census[image_row * width + image_col + threadIdx.x];
					}

					if (image_col + blockDim.x + threadIdx.x < width){
						targ_temp[blockDim.x + threadIdx.x] = left[image_row * width + image_col + blockDim.x + threadIdx.x];
						targ_census_temp[blockDim.x + threadIdx.x] = left_census[image_row * width + image_col + blockDim.x + threadIdx.x];
					}

					__syncthreads();
				}

				float ad_cost, census_cost;

				ad_cost = (fabsf(ref_temp[block_index] - targ_temp[block_index + threadIdx.x]) / 255.0f) * ad_gamma;
				census_cost = (__popcll(ref_census_temp[block_index] ^ targ_census_temp[block_index + threadIdx.x]) / 64.0f) * census_gamma;

				cost += ad_cost + census_cost;

				cost_vol[image_row * width * blockDim.x + image_col * blockDim.x + threadIdx.x] = cost;
			}
		}
	}
}

__global__
void horizontal_aggregation_kernel(float *cost_vol_in, uchar4 *arm_vol, float *cost_vol_out, int width, int height){

	int image_col = blockIdx.x;

	float sum = 0.0f;

	for (int image_row = 0; image_row < height; image_row++){

		uchar4 pixel_arm = arm_vol[image_row * width + image_col];

		int right_limit = image_col + pixel_arm.w;
		int left_limit = image_col - pixel_arm.z - 1;

		float aggregate = cost_vol_in[image_row * width * blockDim.x + right_limit * blockDim.x + threadIdx.x];

		if (left_limit >= 0)
			aggregate -= cost_vol_in[image_row * width * blockDim.x + left_limit * blockDim.x + threadIdx.x];

		sum += aggregate;

		cost_vol_out[image_row * width * blockDim.x + image_col * blockDim.x + threadIdx.x] = sum;
	}
}

__global__
void vertical_aggregation_kernel(float *cost_vol_in, uchar4 *arm_vol, float *cost_vol_out, unsigned short *disp_im, int width, int height){

	int image_row = blockIdx.y;

	__shared__ unsigned int reduce_cache[32];
	__shared__ float cost_cache[256];

	

	for (int image_col = 0; image_col < width; image_col++){

		uchar4 pix_arm = arm_vol[image_row * width + image_col];

		int down_lim = image_row + pix_arm.y;
		int up_lim = image_row - pix_arm.x - 1;

		float aggregate = cost_vol_in[down_lim * width * blockDim.x + image_col * blockDim.x + threadIdx.x];

		if (up_lim >= 0)
			aggregate -= cost_vol_in[up_lim * width * blockDim.x + image_col * blockDim.x + threadIdx.x];

		//cost_vol_out[image_row * width * blockDim.x + image_col * blockDim.x + threadIdx.x] = aggregate;

		cost_cache[threadIdx.x] = aggregate;
		//Find the minimum

		unsigned int min_cost = (((unsigned int)(aggregate * 10000)) << 8) | threadIdx.x;
		unsigned int temp_min_cost = 0;
		

		int lane = threadIdx.x % 32;
		int wid = threadIdx.x / 32;

		temp_min_cost = __shfl_down(min_cost, 16);
		min_cost = min(min_cost, temp_min_cost);

		temp_min_cost = __shfl_down(min_cost, 8);
		min_cost = min(min_cost, temp_min_cost);

		temp_min_cost = __shfl_down(min_cost, 4);
		min_cost = min(min_cost, temp_min_cost);

		temp_min_cost = __shfl_down(min_cost, 2);
		min_cost = min(min_cost, temp_min_cost);

		temp_min_cost = __shfl_down(min_cost, 1);
		min_cost = min(min_cost, temp_min_cost);

		if (lane == 0) reduce_cache[wid] = min_cost;

		__syncthreads();

		min_cost = (threadIdx.x < blockDim.x / 32) ? reduce_cache[lane] : UINT_MAX;

		if (wid == 0){

			temp_min_cost = __shfl_down(min_cost, 4);
			min_cost = min(min_cost, temp_min_cost);

			temp_min_cost = __shfl_down(min_cost, 2);
			min_cost = min(min_cost, temp_min_cost);

			temp_min_cost = __shfl_down(min_cost, 1);
			min_cost = min(min_cost, temp_min_cost);

		}

		if (threadIdx.x == 0){
			unsigned short disp = (unsigned short)((min_cost & 0x000000FF));
			if (disp >= 1 && disp < blockDim.x - 1)
				disp_im[image_row * width + image_col] = (unsigned short)((disp + ((cost_cache[disp + 1] - cost_cache[disp - 1]) / (2 * (-cost_cache[disp + 1] - cost_cache[disp - 1] + 2 * cost_cache[disp])))) * 256.0f);
			else
				disp_im[image_row * width + image_col] = disp << 8;
		}
	}
}

__global__
void consistency_check_kernel(hipTextureObject_t left_disp_im, hipTextureObject_t right_disp_im, unsigned short *output_disp_im, int disparity_tolerance, int width, int height){

	int col_to_access = blockIdx.x * blockDim.x + threadIdx.x;
	int row_to_access = blockIdx.y * blockDim.y + threadIdx.y;

	if (row_to_access < height && col_to_access < width){
		unsigned short disp = tex2D<unsigned short>(left_disp_im, col_to_access, row_to_access);
		unsigned short to_check = tex2D<unsigned short>(right_disp_im, col_to_access - (disp >> 8), row_to_access);

		output_disp_im[row_to_access * width + col_to_access] = (abs(disp - to_check) <= disparity_tolerance * 256) ? disp : OUTLIER;
	}
}

__global__
void horizontal_voting_kernel(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned short *output_disp, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){

		int sums[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
		int eligible_votes = 0;
		int no_of_votes = 0;

		//Load arm data
		uchar4 pix_arm = arm_vol[image_col + image_row * width];

		//Check disp value
		int disp_value = tex2D<unsigned short>(input_disp, image_col, image_row);

		if (disp_value == OUTLIER){
			for (int pix_iter = -pix_arm.z; pix_iter <= pix_arm.w; pix_iter++){
				int disp_val = tex2D<unsigned short>(input_disp, image_col + pix_iter, image_row);
				if (disp_val != OUTLIER){
					sums[0] += ((disp_val & 1) != 0);
					sums[1] += ((disp_val & 2) != 0);
					sums[2] += ((disp_val & 4) != 0);
					sums[3] += ((disp_val & 8) != 0);
					sums[4] += ((disp_val & 16) != 0);
					sums[5] += ((disp_val & 32) != 0);
					sums[6] += ((disp_val & 64) != 0);
					sums[7] += ((disp_val & 128) != 0);
					sums[8] += ((disp_val & 256) != 0);
					sums[9] += ((disp_val & 512) != 0);
					sums[10] += ((disp_val & 1024) != 0);
					sums[11] += ((disp_val & 2048) != 0);
					sums[12] += ((disp_val & 4096) != 0);
					sums[13] += ((disp_val & 8192) != 0);
					sums[14] += ((disp_val & 16384) != 0);
					sums[15] += ((disp_val & 32768) != 0);
					eligible_votes++;
				}
				no_of_votes++;
			}
			__syncthreads();

			int majority = eligible_votes * 0.5;
			disp_value = (
				((sums[15] > majority) << 15) +
				((sums[14] > majority) << 14) +
				((sums[13] > majority) << 13) +
				((sums[12] > majority) << 12) +
				((sums[11] > majority) << 11) +
				((sums[10] > majority) << 10) +
				((sums[9] > majority) << 9) +
				((sums[8] > majority) << 8) +
				((sums[7] > majority) << 7) +
				((sums[6] > majority) << 6) +
				((sums[5] > majority) << 5) +
				((sums[4] > majority) << 4) +
				((sums[3] > majority) << 3) +
				((sums[2] > majority) << 2) +
				((sums[1] > majority) << 1) +
				((sums[0] > majority) << 0));
			disp_value = (eligible_votes > no_of_votes * 0.35f) ? disp_value : OUTLIER;
		}

		output_disp[image_col + image_row * width] = disp_value;
	}
}

__global__
void vertical_voting_kernel(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned short *output_disp, int width, int height){
	int image_row = blockIdx.y * blockDim.y + threadIdx.y;
	int image_col = blockIdx.x * blockDim.x + threadIdx.x;

	if (image_row < height && image_col < width){

		int sums[16] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };
		int eligible_votes = 0;
		int no_of_votes = 0;

		//Load arm data
		uchar4 pix_arm = arm_vol[image_col + image_row * width];

		//Check disp value
		int disp_value = tex2D<unsigned short>(input_disp, image_col, image_row);

		if (disp_value == OUTLIER){
			for (int pix_iter = -pix_arm.x; pix_iter <= pix_arm.y; pix_iter++){
				int disp_val = tex2D<unsigned char>(input_disp, image_col, image_row + pix_iter);
				if (disp_val != OUTLIER){
					sums[0] += ((disp_val & 1) != 0);
					sums[1] += ((disp_val & 2) != 0);
					sums[2] += ((disp_val & 4) != 0);
					sums[3] += ((disp_val & 8) != 0);
					sums[4] += ((disp_val & 16) != 0);
					sums[5] += ((disp_val & 32) != 0);
					sums[6] += ((disp_val & 64) != 0);
					sums[7] += ((disp_val & 128) != 0);
					sums[8] += ((disp_val & 256) != 0);
					sums[9] += ((disp_val & 512) != 0);
					sums[10] += ((disp_val & 1024) != 0);
					sums[11] += ((disp_val & 2048) != 0);
					sums[12] += ((disp_val & 4096) != 0);
					sums[13] += ((disp_val & 8192) != 0);
					sums[14] += ((disp_val & 16384) != 0);
					sums[15] += ((disp_val & 32768) != 0);
					eligible_votes++;
				}
				no_of_votes++;
			}
			__syncthreads();

			int majority = eligible_votes * 0.5;
			disp_value = (
				((sums[15] > majority) << 15) +
				((sums[14] > majority) << 14) +
				((sums[13] > majority) << 13) +
				((sums[12] > majority) << 12) +
				((sums[11] > majority) << 11) +
				((sums[10] > majority) << 10) +
				((sums[9] > majority) << 9) +
				((sums[8] > majority) << 8) +
				((sums[7] > majority) << 7) +
				((sums[6] > majority) << 6) +
				((sums[5] > majority) << 5) +
				((sums[4] > majority) << 4) +
				((sums[3] > majority) << 3) +
				((sums[2] > majority) << 2) +
				((sums[1] > majority) << 1) +
				((sums[0] > majority) << 0));
			disp_value = (eligible_votes > no_of_votes * 0.35f) ? disp_value : OUTLIER;
		}

		output_disp[image_col + image_row * width] = disp_value;
	}
}


__global__
void median_filter_kernel(unsigned short *d_in, unsigned short *d_out, int nx, int ny)
{
	int tx = threadIdx.x, ty = threadIdx.y;

	// guards: is at boundary?
	bool is_x_top = (tx == 0), is_x_bot = (tx == BLOCK_X - 1);
	bool is_y_top = (ty == 0), is_y_bot = (ty == BLOCK_Y - 1);

	__shared__ unsigned short smem[BLOCK_X + 2][BLOCK_Y + 2];
	// clear out shared memory (zero padding)
	if (is_x_top)           SMEM(tx - 1, ty) = 0;
	else if (is_x_bot)      SMEM(tx + 1, ty) = 0;
	if (is_y_top) {
		SMEM(tx, ty - 1) = 0;
		if (is_x_top)       SMEM(tx - 1, ty - 1) = 0;
		else if (is_x_bot)  SMEM(tx + 1, ty - 1) = 0;
	}
	else if (is_y_bot) {
		SMEM(tx, ty + 1) = 0;
		if (is_x_top)       SMEM(tx - 1, ty + 1) = 0;
		else if (is_x_bot)  SMEM(tx + 1, ty + 1) = 0;
	}

	// guards: is at boundary and still more image?
	int x = blockIdx.x * blockDim.x + tx;
	int y = blockIdx.y * blockDim.y + ty;
	is_x_top &= (x > 0); is_x_bot &= (x < nx - 1);
	is_y_top &= (y > 0); is_y_bot &= (y < ny - 1);

	// each thread pulls from image
	SMEM(tx, ty) = IN(x, y); // self
	if (is_x_top)           SMEM(tx - 1, ty) = IN(x - 1, y);
	else if (is_x_bot)      SMEM(tx + 1, ty) = IN(x + 1, y);
	if (is_y_top) {
		SMEM(tx, ty - 1) = IN(x, y - 1);
		if (is_x_top)       SMEM(tx - 1, ty - 1) = IN(x - 1, y - 1);
		else if (is_x_bot)  SMEM(tx + 1, ty - 1) = IN(x + 1, y - 1);
	}
	else if (is_y_bot) {
		SMEM(tx, ty + 1) = IN(x, y + 1);
		if (is_x_top)       SMEM(tx - 1, ty + 1) = IN(x - 1, y + 1);
		else if (is_x_bot)  SMEM(tx + 1, ty + 1) = IN(x + 1, y + 1);
	}
	__syncthreads();

	// pull top six from shared memory
	unsigned short v[6] = { SMEM(tx - 1, ty - 1), SMEM(tx, ty - 1), SMEM(tx + 1, ty - 1),
		SMEM(tx - 1, ty), SMEM(tx, ty), SMEM(tx + 1, ty) };

	// with each pass, remove min and max values and add new value
	mnmx6(v[0], v[1], v[2], v[3], v[4], v[5]);
	v[5] = SMEM(tx - 1, ty + 1); // add new contestant
	mnmx5(v[1], v[2], v[3], v[4], v[5]);
	v[5] = SMEM(tx, ty + 1);
	mnmx4(v[2], v[3], v[4], v[5]);
	v[5] = SMEM(tx + 1, ty + 1);
	mnmx3(v[3], v[4], v[5]);

	// pick the middle one
	d_out[y*nx + x] = v[4];
}


/////////////////////////////////////////////////////////////////////////////Stubs/////////////////////////////////////////////////////////////////////////////

void census_transform(hipTextureObject_t input_im, unsigned long long int *output_census, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	census_transform_kernel << <blocks, threads >> >(input_im, output_census, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Census transform failed.");
#endif
}

void cross_construct(hipTextureObject_t input_im, uchar4 *arm_vol, int arm_length, int max_arm_length, int arm_threshold, int strict_arm_threshold, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	cross_construct_kernel << <blocks, threads >> >(input_im, arm_vol, arm_length, max_arm_length, arm_threshold, strict_arm_threshold, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Cross construct failed.");
#endif
}

void match(unsigned char *left, unsigned char *right,
	unsigned long long int *left_census, unsigned long long int *right_census, float *cost_vol_temp_a, float *cost_vol_temp_b, uchar4 *arm_vol,
	unsigned short *disp_im, float gamma, float census_gamma, bool left_to_right, int width, int height, int max_disparity, hipStream_t stream){

	dim3 b(1, height); dim3 t(max_disparity);
	size_t mem_sz = t.x * (sizeof(unsigned long long int) + sizeof(unsigned char)) * 3;
	cost_initialization_kernel << <b, t, mem_sz >> >(left, right, left_census, right_census, (float*)cost_vol_temp_a, gamma, census_gamma, left_to_right, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Cost initialization failed.");
#endif

	b = dim3(width);
	t = dim3(max_disparity);
	horizontal_aggregation_kernel << < b, t >> > ((float*)cost_vol_temp_a, arm_vol, (float*)cost_vol_temp_b, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal aggregation failed.");
#endif

	b = dim3(1, height);
	t = dim3(max_disparity);
	vertical_aggregation_kernel << < b, t >> > ((float*)cost_vol_temp_b, arm_vol, (float*)cost_vol_temp_a, disp_im, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal aggregation failed.");
#endif
}

void check_consistency(hipTextureObject_t left_disp_im, hipTextureObject_t right_disp_im, unsigned short *output_disp_im, int disparity_tolerance, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	consistency_check_kernel << <blocks, threads >> >(left_disp_im, right_disp_im, output_disp_im, disparity_tolerance, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Consistency check failed.");
#endif
}

void horizontal_voting(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned short *output_disp, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	horizontal_voting_kernel << <blocks, threads >> >(input_disp, arm_vol, output_disp, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Horizontal voting failed.");
#endif
}

void vertical_voting(hipTextureObject_t input_disp, uchar4 *arm_vol, unsigned short *output_disp, int width, int height, hipStream_t stream){
	dim3 threads(16, 16);
	dim3 blocks(DIVIDE_UP(width, threads.x), DIVIDE_UP(height, threads.y));

	vertical_voting_kernel << <blocks, threads >> >(input_disp, arm_vol, output_disp, width, height);
#ifdef KERN_DEB
	SAFE_CALL(hipDeviceSynchronize(), "Vertical voting failed.");
#endif
}

void median_filter(unsigned short *input_disp, unsigned short *output_disp, int width, int height){
	dim3 blocks(width / BLOCK_X, height / BLOCK_Y);
	dim3 threads(BLOCK_X, BLOCK_Y);
	median_filter_kernel << <blocks, threads >> >(input_disp, output_disp, width, height);
}


